#include "hip/hip_runtime.h"

#include <cstdio>
#include "device_reduce_atomic.h"
#include "device_reduce_block_atomic.h"
#include "device_reduce_warp_atomic.h"
#include "device_reduce_stable.h"
#include "vector_functions.h"
#include "cub/hipcub/hipcub.hpp"

//Tangram generated code
//#include "Tangram_DistTile_DistTile_S_S_S.h"
//#include "Tangram_DistTile_DistTile_S_S_S.h"

#include "Tangram_DistTileAtomic_DistTileAtomic_S.h"
#include "Tangram_DistTileAtomic_Vs.h"
#include "Tangram_DistTileAtomic_Va1s.h"
#include "Tangram_DistTileAtomic_Va2.h"
#include "Tangram_DistTileAtomic_Va1.h"
#include "Tangram_DistTileAtomic_V.h"
#include "Tangram_DistTile_Va1s_Va1s.h"
#include "Tangram_DistTile_Va1s_Va1.h"
#include "Tangram_DistTile_Va1s_Va2.h"
#include "Tangram_DistTile_Va1s_Vs.h"
#include "Tangram_DistTile_Va1s_V.h"
#include "Tangram_DistTile_Va2_Va1s.h"
#include "Tangram_DistTile_Va2_Va1.h"
#include "Tangram_DistTile_Va2_Va2.h"
#include "Tangram_DistTile_Va2_Vs.h"
#include "Tangram_DistTile_Va2_V.h"
#include "Tangram_DistTileAtomic_DistStride_S_Va1s.h"
#include "Tangram_DistTileAtomic_DistStride_S_Va2.h"
#include "Tangram_DistTileAtomic_DistStride_S_Va1.h"
#include "Tangram_DistTileAtomic_DistStride_S_Vs.h"
#include "Tangram_DistTileAtomic_DistStride_S_V.h"

#define cudaCheckError() {                                          \
  hipError_t e=hipGetLastError();                                  \
  if(e!=hipSuccess) {                                               \
  printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
  exit(0); \
  }                                                                  \
}

void RunTest(char* label, void (*fptr)(int* in, int* out, int N), int N, int REPEAT, int* src, int checksum) {
  int *in, *out;
  
  //allocate a buffer that is at least large enough that we can ensure it doesn't just sit in l2.
  int MIN_SIZE=4*1024*1024;
  int size=max(int(sizeof(int)*N),MIN_SIZE);
  
  //compute mod base for picking the correct buffer
  int mod=size/(N*sizeof(int));
  hipEvent_t start,stop;
  hipMalloc(&in,size);
  hipMalloc(&out,sizeof(int)*1024);  //only stable version needs multiple elements, all others only need 1
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaCheckError();

  hipMemcpy(in,src,N*sizeof(int),hipMemcpyHostToDevice);
  
  //warm up
  fptr(in,out,N);

  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);

  for(int i=0;i<REPEAT;i++) {
    //iterate through different buffers
    int o=i%mod;
    fptr(in+o*N,out,N);
  }
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();

  float time_ms;
  hipEventElapsedTime(&time_ms,start,stop);
  float time_s=time_ms/(float)1e3;

  float GB=(float)N*sizeof(int)*REPEAT;
  float GBs=GB/time_s/(float)1e9;

  int sum;
  hipMemcpy(&sum,out,sizeof(int),hipMemcpyDeviceToHost);
  cudaCheckError();

  char *valid;
  if(sum==checksum) 
    valid="CORRECT";
  else
    valid="INCORRECT";

  printf("%s: %s, Time: %f s, GB/s: %f\n", label, valid, time_s, GBs); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(in);
  hipFree(out);
  cudaCheckError();
}

void RunTestCub(char* label, int N, int REPEAT, int* src, int checksum) {
  int *in, *out;
  hipEvent_t start,stop;
  
  hipMalloc(&in,sizeof(int)*N);
  hipMalloc(&out,sizeof(int)*1024);  //only stable version needs multiple elements, all others only need 1
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaCheckError();

  hipMemcpy(in,src,N*sizeof(int),hipMemcpyHostToDevice);

  size_t temp_storage_bytes;
  int* temp_storage=NULL;
  hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, in, out, N, hipcub::Sum(), 0);
  hipMalloc(&temp_storage,temp_storage_bytes);

  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);

  for(int i=0;i<REPEAT;i++) {
    hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, in, out, N, hipcub::Sum(), 0);
  }
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();

  float time_ms;
  hipEventElapsedTime(&time_ms,start,stop);
  float time_s=time_ms/(float)1e3;

  float GB=(float)N*sizeof(int)*REPEAT;
  float GBs=GB/time_s/(float)1e9;

  int sum;
  hipMemcpy(&sum,out,sizeof(int),hipMemcpyDeviceToHost);
  cudaCheckError();

  char *valid;
  if(sum==checksum) 
    valid="CORRECT";
  else
    valid="INCORRECT";

  printf("%s: %s, Time: %f s, GB/s: %f, Sum: %d\n", label, valid, time_s, GBs, sum); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(in);
  hipFree(out);
  hipFree(temp_storage);
  cudaCheckError();
}


void RunTestTangram(char* label, void (*fptr)(int* in, int* out, int N, int blockNum, int threadNum), int N, int REPEAT, int* src, int checksum, int blockNum, int threadNum) {
  int *in, *out;
  //allocate a buffer that is at least large enough that we can ensure it doesn't just sit in l2.
  int MIN_SIZE=4*1024*1024;
  int size=max(int(sizeof(int)*N),MIN_SIZE);
  
  //compute mod base for picking the correct buffer
  int mod=size/(N*sizeof(int));
  hipEvent_t start,stop;
  hipMalloc(&in,size);
  hipMalloc(&out,sizeof(int)*1024);  //only stable version needs multiple elements, all others only need 1
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaCheckError();

  hipMemcpy(in,src,N*sizeof(int),hipMemcpyHostToDevice);
  

  
  fptr(in,out,N, blockNum, threadNum);

  hipMemset(out,0,sizeof(int));

  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);

  for(int i=0;i<REPEAT;i++) {
    //iterate through different buffers
    //int o=i%mod;
    fptr(in,out,N, blockNum, threadNum);
  }
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();

  float time_ms;
  hipEventElapsedTime(&time_ms,start,stop);
  float time_s=time_ms/(float)1e3;

  float GB=(float)N*sizeof(int)*REPEAT;
  float GBs=GB/time_s/(float)1e9;

  int sum;
  hipMemcpy(&sum,out,sizeof(int),hipMemcpyDeviceToHost);
  cudaCheckError();

  char *valid;
  if(sum==checksum) 
    valid="CORRECT";
  else
    valid="INCORRECT";

  printf("%s: %s, Time: %f s, GB/s: %f, Sum: %d\n", label, valid, time_s, GBs, sum); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(in);
  hipFree(out);
  cudaCheckError();
}



void RunTestTangram_2(char* label, void (*fptr)(int* in, int* out, int N, int blockNum), int N, int REPEAT, int* src, int checksum, int blockNum) {
  int *in, *out;
  //int sum = 0;  
  //allocate a buffer that is at least large enough that we can ensure it doesn't just sit in l2.
  int MIN_SIZE=4*1024*1024;
  int size=max(int(sizeof(int)*N),MIN_SIZE);
  
  //compute mod base for picking the correct buffer
  int mod=size/(N*sizeof(int));
  hipEvent_t start,stop;
  hipMalloc(&in,size);
  hipMalloc(&out,sizeof(int)*1024);  //only stable version needs multiple elements, all others only need 1
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaCheckError();

  hipMemcpy(in,src,N*sizeof(int),hipMemcpyHostToDevice);
  

  //warm up
  fptr(in,out,N,blockNum);

  hipMemset(out,0,sizeof(int));


  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);

  for(int i=0;i<REPEAT;i++) {
    //iterate through different buffers
    //int o=i%mod;
    fptr(in/*+o*N*/,out,N,blockNum);
  }
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();

  float time_ms;
  hipEventElapsedTime(&time_ms,start,stop);
  float time_s=time_ms/(float)1e3;

  float GB=(float)N*sizeof(int)*REPEAT;
  float GBs=GB/time_s/(float)1e9;

  int sum;
  hipMemcpy(&sum,out,sizeof(int),hipMemcpyDeviceToHost);
  cudaCheckError();

  char *valid;
  if(sum==checksum) 
    valid="CORRECT";
  else
    valid="INCORRECT";

  printf("%s: %s, Time: %f s, GB/s: %f, Sum: %d\n", label, valid, time_s, GBs, sum); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(in);
  hipFree(out);
  cudaCheckError();
}


int main(int argc, char** argv)
{
  if(argc<3) {
    printf("Usage: ./reduce num_elems repeat blockNum(optional) threadNum(optional)\n");
    exit(0);
  }
  int NUM_ELEMS=atoi(argv[1]);
  int REPEAT=atoi(argv[2]);

  int NUM_BLOCKS = 128;
  int NUM_THREADS = 512;
  int NUM_BLOCKS_VEC = 1024;
  if(argc == 5){
    NUM_BLOCKS = atoi(argv[3]);
    NUM_THREADS = atoi(argv[4]);
  }
  else if(argc == 6){
   NUM_BLOCKS = atoi(argv[3]);
   NUM_THREADS = atoi(argv[4]);
   NUM_BLOCKS_VEC = atoi(argv[5]);
  }

  printf("NUM_ELEMS: %d, REPEAT: %d, NUM_BLOCKS: %d, NUM_THREADS: %d, NUM_BLOCKS_VEC: %d\n", NUM_ELEMS, REPEAT, NUM_BLOCKS, NUM_THREADS, NUM_BLOCKS_VEC);

  int* vals=(int*)malloc(NUM_ELEMS*sizeof(int));
  int checksum =0;
  for(int i=0;i<NUM_ELEMS;i++) {
    vals[i]= 1;//rand()%4;
    checksum+=vals[i];
  }

  //RunTest("device_reduce_atomic", device_reduce_atomic,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_atomic_vector2", device_reduce_atomic_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_atomic_vector4", device_reduce_atomic_vector4,NUM_ELEMS,REPEAT,vals,checksum);
  
  //RunTest("device_reduce_warp_atomic",device_reduce_warp_atomic,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_warp_atomic_vector2",device_reduce_warp_atomic_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_warp_atomic_vector4",device_reduce_warp_atomic_vector4,NUM_ELEMS,REPEAT,vals,checksum);
  
  //RunTest("device_reduce_block_atomic",device_reduce_block_atomic,NUM_ELEMS,REPEAT,vals,checksum);
  
  //RunTest("device_reduce_stable",device_reduce_stable,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_stable_vector2",device_reduce_stable_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_stable_vector4",device_reduce_stable_vector4,NUM_ELEMS,REPEAT,vals,checksum);

  RunTestCub("device_reduce_cub",NUM_ELEMS,REPEAT,vals,checksum);

  //RunTestTangram("tangram_dTile_dTile_S_S_S", tangram_dTile_dTile_S_S_S, NUM_ELEMS,REPEAT,vals,checksum);
   
  RunTestTangram_2("tangram_dTileAtom_Va1s", tangram_dTileAtom_Va1s, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTileAtom_Va2", tangram_dTileAtom_Va2, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTileAtom_Va1", tangram_dTileAtom_Va1, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTileAtom_Vs", tangram_dTileAtom_Vs, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTileAtom_V", tangram_dTileAtom_V, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
#if 0
  RunTestTangram_2("tangram_dTile_Va1s_Va1s", tangram_dTile_Va1s_Va1s, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTile_Va1s_Va1", tangram_dTile_Va1s_Va1, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTile_Va1s_Va2", tangram_dTile_Va1s_Va2, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTile_Va1s_Vs", tangram_dTile_Va1s_Vs, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTile_Va1s_V", tangram_dTile_Va1s_V, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTile_Va2_Va1s", tangram_dTile_Va2_Va1s, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTile_Va2_Va1", tangram_dTile_Va2_Va1, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTile_Va2_Va2", tangram_dTile_Va2_Va2, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTile_Va2_Vs", tangram_dTile_Va2_Vs, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
  RunTestTangram_2("tangram_dTile_Va2_V", tangram_dTile_Va2_V, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS_VEC);
#endif

  RunTestTangram("tangram_dTileAtom_dStride_S_Va1s", tangram_dTileAtom_dStride_S_Va1s, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS, NUM_THREADS);
  RunTestTangram("tangram_dTileAtom_dStride_S_Va2", tangram_dTileAtom_dStride_S_Va2, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS, NUM_THREADS);
  RunTestTangram("tangram_dTileAtom_dStride_S_Va1", tangram_dTileAtom_dStride_S_Va1, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS, NUM_THREADS);
  RunTestTangram("tangram_dTileAtom_dStride_S_Vs", tangram_dTileAtom_dStride_S_Vs, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS, NUM_THREADS);
  RunTestTangram("tangram_dTileAtom_dStride_S_V", tangram_dTileAtom_dStride_S_V, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS, NUM_THREADS);
  RunTestTangram("tangram_dTileAtom_dStrideAtom_S", tangram_dTileAtom_dStrideAtom_S, NUM_ELEMS,REPEAT,vals,checksum, NUM_BLOCKS, NUM_THREADS);
  
  free(vals);

}
